#include "hip/hip_runtime.h"
#include "asm.h"

#define MAX_QUERY_LENGTH 1500
#define SHARED_SIZE 4000

#define BLOCK_NUM 60
#define THREAD_NUM 512

#define TEST_SEQ 39

#define INF 1e10

#define dist(x,y) (fabs(x-y))

__constant__ float query_gpu[MAX_QUERY_LENGTH];

float* Asm(float *query,int qline,int seq_num,float *all_seq,int all_length,int *seq_length,int *accu_length,float r)
{
    float t1,t2;
    float *all_seq_gpu;
    int *seq_length_gpu;
    int *accu_length_gpu;

    if(qline<=MAX_QUERY_LENGTH)
    {
        cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(query_gpu),query,qline*sizeof(float)));
    }
    else//limit the length of query to 2K
    {
        cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(query_gpu),query,MAX_QUERY_LENGTH*sizeof(float)));
        qline=MAX_QUERY_LENGTH;
    }
    //printf("query length is %d\n",qline);


    cutilSafeCall(hipMalloc((void **)&all_seq_gpu,sizeof(float)*all_length));
    cutilSafeCall(hipMemcpy(all_seq_gpu,all_seq,all_length*sizeof(float),hipMemcpyHostToDevice));

    cutilSafeCall(hipMalloc((void **)&seq_length_gpu,sizeof(int)*seq_num));
    cutilSafeCall(hipMemcpy(seq_length_gpu,seq_length,seq_num*sizeof(int),hipMemcpyHostToDevice));

    cutilSafeCall(hipMalloc((void **)&accu_length_gpu,sizeof(int)*seq_num));
    cutilSafeCall(hipMemcpy(accu_length_gpu,accu_length,seq_num*sizeof(int),hipMemcpyHostToDevice));

    //malloc space for the variables used in asm
    float *cost_gpu;
    int *start_gpu;
    int *x_gpu;
    int *y_gpu;
    cutilSafeCall(hipMalloc((void **)&cost_gpu,sizeof(float)*qline*seq_num*3));
    cutilSafeCall(hipMalloc((void **)&start_gpu,sizeof(int)*qline*seq_num*3));
    cutilSafeCall(hipMalloc((void **)&x_gpu,sizeof(int)*qline*seq_num*3));
    cutilSafeCall(hipMalloc((void **)&y_gpu,sizeof(int)*qline*seq_num*3));

    int *left_bound_gpu,*right_bound_gpu;
    cutilSafeCall(hipMalloc((void **)&left_bound_gpu,sizeof(int)*qline));
    cutilSafeCall(hipMalloc((void **)&right_bound_gpu,sizeof(int)*qline));


    float *small_gpu;
    float *small;
    cutilSafeCall(hipMalloc((void **)&small_gpu,sizeof(float)*seq_num));
    small=(float*)malloc(sizeof(float)*seq_num);

    bound<<<BLOCK_NUM,THREAD_NUM/4>>>(qline,r,left_bound_gpu,right_bound_gpu);

    /*
    float *dist,*dist_gpu;
    dist=(float*)malloc(sizeof(float)*seq_length[TEST_SEQ]);
    cutilSafeCall(hipMalloc((void **)&dist_gpu,sizeof(float)*seq_length[TEST_SEQ]));

    float *diagonal,*diagonal_gpu;
    diagonal=(float*)malloc(sizeof(float)*qline);
    cutilSafeCall(hipMalloc((void **)&diagonal_gpu,sizeof(float)*qline));
    */

    t1=clock();
    _asm<<<seq_num,THREAD_NUM,sizeof(int)*qline*2>>>(qline,all_seq_gpu,seq_length_gpu,
            accu_length_gpu,cost_gpu,start_gpu,x_gpu,y_gpu,
            left_bound_gpu,right_bound_gpu,small_gpu);
    hipDeviceSynchronize();
    t2=clock();
    //printf("cal time is %lf\n",(t2-t1)/CLOCKS_PER_SEC);

    cutilSafeCall(hipMemcpy(small,small_gpu,sizeof(float)*seq_num,hipMemcpyDeviceToHost));

    for(int i=0;i<seq_num;i++)
    {
        //printf("%d:%lf\n",i+1,small[i]);
    }

    /*
    cutilSafeCall(hipMemcpy(dist,dist_gpu,sizeof(float)*seq_length[TEST_SEQ],hipMemcpyDeviceToHost));

    for(int i=0;i<seq_length[TEST_SEQ];i++)
    {
        printf("%d:%f\n",i+1,dist[i]);
    }

    cutilSafeCall(hipMemcpy(diagonal,diagonal_gpu,sizeof(float)*qline,hipMemcpyDeviceToHost));

    printf("diagonal is:\n");
    for(int i=0;i<qline;i++)
    {
        printf("%d:%f\n",i+1,diagonal[i]);
    }
    */

    return small;
}

__global__ static void bound(int qline,float r,int *left_bound_gpu,int *right_bound_gpu)
{
    int tid=threadIdx.x;
    int bid=blockIdx.x;

    int tsize=blockDim.x;
    int bsize=gridDim.x;

    float warp_width=qline*r;

    for(int i=tid+tsize*bid;i<qline;i+=bsize*tsize)
    {
        left_bound_gpu[i]=(int)((i+1)*0.2f+warp_width); 
        right_bound_gpu[i]=(int)((i+1)*0.25f+warp_width); 
    }
}

__device__ inline static float valid_pos(int left_bound,int right_bound, int x,int y,int kind)
{
    switch(kind)
    {
        case 1:
            y++;
            break;
        case 2:
            x++;
            break;
        default:
            break;
    }
    if(x>y&&x-y<=left_bound) return 0;
    else if(x<y&&y-x<=right_bound) return 0;
    else if(x==y) return 0;
    else return INF;
}

__device__ inline static float min(float a,float b,int *index)
{
    if(a<b)
    {
        *index=1;
        return a;
    }
    else
    {
        *index=2;
        return b;
    }
}

__global__ static void _asm(int qline,float *all_seq_gpu,
        int *seq_length_gpu,int *accu_length_gpu,
        float *cost_gpu,int *start_gpu,int *x_gpu,int *y_gpu,
        int *left_bound_gpu,int *right_bound_gpu,float *small_gpu)
{
    int tid=threadIdx.x;
    int bid=blockIdx.x;   
    int tsize=blockDim.x;

    __shared__ int seq_start;
    __shared__ int seq_length;

    extern __shared__ int left_right_bound[];

    __shared__ int out_start;//mark the start position of the cal row of every sequence
    __shared__ int inner_start;//mark the position of pprev,prev and cur

    __shared__ int pprev_start;
    __shared__ int prev_start;
    __shared__ int cur_start;

    __shared__ int start;
    __shared__ int end;

    __shared__ float small;
    //__shared__ int start_pos,end_pos;
    __shared__ float scale_factor;

    if(tid==0)
    {
        seq_start=accu_length_gpu[bid];
        seq_length=seq_length_gpu[bid];
        
        out_start=bid*qline*3;//mark the start position of the cal row 
        inner_start=0;

        pprev_start=0;
        prev_start=(inner_start+1)*qline;
        cur_start=(inner_start+2)*qline;
    }

    __syncthreads();

    for(int i=tid;i<3*qline;i+=tsize)
    {
        cost_gpu[out_start+i]=INF;
        start_gpu[out_start+i]=0;
    }
    __syncthreads();

    for(int i=tid;i<qline;i+=tsize)
    {
        left_right_bound[i]=left_bound_gpu[i];
        left_right_bound[qline+i]=right_bound_gpu[i]; 
    }

    __syncthreads();

    if(tid==0)
    {
        cost_gpu[out_start+pprev_start+qline-1]=dist(query_gpu[0],all_seq_gpu[seq_start]);
        cost_gpu[out_start+prev_start+qline-2]=dist(query_gpu[1],all_seq_gpu[seq_start])+cost_gpu[out_start+pprev_start+qline-1];
        cost_gpu[out_start+prev_start+qline-1]=dist(query_gpu[0],all_seq_gpu[seq_start+1]);

        start_gpu[out_start+pprev_start+qline-1]=1;
        start_gpu[out_start+prev_start+qline-2]=1;
        start_gpu[out_start+prev_start+qline-1]=2;

        x_gpu[out_start+pprev_start+qline-1]=1;
        y_gpu[out_start+pprev_start+qline-1]=1;

        x_gpu[out_start+prev_start+qline-2]=2;
        y_gpu[out_start+prev_start+qline-2]=1;

        x_gpu[out_start+prev_start+qline-1]=1;
        y_gpu[out_start+prev_start+qline-1]=1;

        start=qline-3;
        end=qline-2;

        small=INF;
    }

    __syncthreads();


    for(int i=3;i<qline;i++)
    {
        for(int j=start+tid;j<qline-1;j+=tsize)
        {
            float v1=valid_pos(left_right_bound[i-1-(j-start)],left_right_bound[qline+i-1-(j-start)],x_gpu[out_start+prev_start+j],y_gpu[out_start+prev_start+j],1); 
            float v2=valid_pos(left_right_bound[i-1-(j-start)],left_right_bound[qline+i-1-(j-start)],x_gpu[out_start+prev_start+j+1],y_gpu[out_start+prev_start+j+1],2); 

            int index1,index2;
            float small1=min(cost_gpu[out_start+prev_start+j]+v1,cost_gpu[out_start+prev_start+j+1]+v2,&index1);
            float small2=min(small1,cost_gpu[out_start+pprev_start+j+1],&index2);

            cost_gpu[out_start+cur_start+j]=dist(query_gpu[i-1-(j-start)],all_seq_gpu[seq_start+j-start])+small2;

            if(index2==2)
            {
                start_gpu[out_start+cur_start+j]=start_gpu[out_start+pprev_start+j+1];
                x_gpu[out_start+cur_start+j]=x_gpu[out_start+pprev_start+j+1]+1;
                y_gpu[out_start+cur_start+j]=y_gpu[out_start+pprev_start+j+1]+1;
            }
            else if(index1==2)
            {
                start_gpu[out_start+cur_start+j]=start_gpu[out_start+prev_start+j+1];
                x_gpu[out_start+cur_start+j]=x_gpu[out_start+prev_start+j+1]+1;
                y_gpu[out_start+cur_start+j]=y_gpu[out_start+prev_start+j+1];
            }
            else
            {
                start_gpu[out_start+cur_start+j]=start_gpu[out_start+prev_start+j];
                x_gpu[out_start+cur_start+j]=x_gpu[out_start+prev_start+j];
                y_gpu[out_start+cur_start+j]=y_gpu[out_start+prev_start+j]+1;
            }
        } 

        if(tid==0)
        {
            cost_gpu[out_start+cur_start+qline-1]=dist(query_gpu[0],all_seq_gpu[seq_start+i-1]);
            start_gpu[out_start+cur_start+qline-1]=i;
            x_gpu[out_start+cur_start+qline-1]=1;
            y_gpu[out_start+cur_start+qline-1]=1;

            inner_start++;
            pprev_start=(inner_start%3)*qline;
            prev_start=((inner_start+1)%3)*qline;
            cur_start=((inner_start+2)%3)*qline;

            start--;
        }

        /*
        __syncthreads();
        if(i==3&&bid==TEST_SEQ)
        {
            for(int k=tid;k<qline;k+=tsize)
            {
                diagonal_gpu[k]=cost_gpu[out_start+prev_start+k];
            }
        }
        */

        __syncthreads();
    }


    for(int i=qline;i<=seq_length;i++)
    {
        for(int j=tid;j<qline-1;j+=tsize)
        {
            float v1=valid_pos(left_right_bound[qline-1-j],left_right_bound[2*qline-1-j],x_gpu[out_start+prev_start+j],y_gpu[out_start+prev_start+j],1); 
            float v2=valid_pos(left_right_bound[qline-1-j],left_right_bound[2*qline-1-j],x_gpu[out_start+prev_start+j+1],y_gpu[out_start+prev_start+j+1],2); 

            int index1,index2;
            float small1=min(cost_gpu[out_start+prev_start+j]+v1,cost_gpu[out_start+prev_start+j+1]+v2,&index1);
            float small2=min(small1,cost_gpu[out_start+pprev_start+j+1],&index2);

            cost_gpu[out_start+cur_start+j]=dist(query_gpu[qline-1-j],all_seq_gpu[seq_start+i-qline+j])+small2;
            if(index2==2)
            {
                start_gpu[out_start+cur_start+j]=start_gpu[out_start+pprev_start+j+1];
                x_gpu[out_start+cur_start+j]=x_gpu[out_start+pprev_start+j+1]+1;
                y_gpu[out_start+cur_start+j]=y_gpu[out_start+pprev_start+j+1]+1;
            }
            else if(index1==2)
            {
                start_gpu[out_start+cur_start+j]=start_gpu[out_start+prev_start+j+1];
                x_gpu[out_start+cur_start+j]=x_gpu[out_start+prev_start+j+1]+1;
                y_gpu[out_start+cur_start+j]=y_gpu[out_start+prev_start+j+1];
            }
            else
            {
                start_gpu[out_start+cur_start+j]=start_gpu[out_start+prev_start+j];
                x_gpu[out_start+cur_start+j]=x_gpu[out_start+prev_start+j];
                y_gpu[out_start+cur_start+j]=y_gpu[out_start+prev_start+j]+1;
            }
        }

        if(tid==0)
        {
            cost_gpu[out_start+cur_start+qline-1]=dist(query_gpu[0],all_seq_gpu[seq_start+i-1]);
            start_gpu[out_start+cur_start+qline-1]=i;
            x_gpu[out_start+cur_start+qline-1]=1;
            y_gpu[out_start+cur_start+qline-1]=1;

            if(cost_gpu[out_start+cur_start]<small)
            {
                scale_factor=(i-qline+2-start_gpu[out_start+cur_start])*1.0f/qline;
                if(scale_factor>=0.8f&&scale_factor<=1.25f)
                {
                    small=cost_gpu[out_start+cur_start];
                    //start_pos=cur_start_gpu[out_start];
                    //end_pos=i-qline+1;
                }
            }

            /*
            if(bid==TEST_SEQ)
            {
                dist_gpu[i-qline]=cost_gpu[out_start+cur_start];
            }
            */

            inner_start++;
            pprev_start=(inner_start%3)*qline;
            prev_start=((inner_start+1)%3)*qline;
            cur_start=((inner_start+2)%3)*qline;
        }

        __syncthreads();
    }

    for(int i=seq_length+1;i<qline+seq_length;i++)
    {
        for(int j=tid;j<=end;j+=tsize)
        {
            float v1=valid_pos(left_right_bound[qline-1-j],left_right_bound[2*qline-1-j],x_gpu[out_start+prev_start+j],y_gpu[out_start+prev_start+j],1); 
            float v2=valid_pos(left_right_bound[qline-1-j],left_right_bound[2*qline-1-j],x_gpu[out_start+prev_start+j+1],y_gpu[out_start+prev_start+j+1],2); 

            int index1,index2;
            float small1=min(cost_gpu[out_start+prev_start+j]+v1,cost_gpu[out_start+prev_start+j+1]+v2,&index1);
            float small2=min(small1,cost_gpu[out_start+pprev_start+j+1],&index2);

            cost_gpu[out_start+cur_start+j]=dist(query_gpu[qline-1-j],all_seq_gpu[seq_start+i-qline+j])+small2;
            if(index2==2)
            {
                start_gpu[out_start+cur_start+j]=start_gpu[out_start+pprev_start+j+1];
                x_gpu[out_start+cur_start+j]=x_gpu[out_start+pprev_start+j+1]+1;
                y_gpu[out_start+cur_start+j]=y_gpu[out_start+pprev_start+j+1]+1;
            }
            else if(index1==2)
            {
                start_gpu[out_start+cur_start+j]=start_gpu[out_start+prev_start+j+1];
                x_gpu[out_start+cur_start+j]=x_gpu[out_start+prev_start+j+1]+1;
                y_gpu[out_start+cur_start+j]=y_gpu[out_start+prev_start+j+1];
            }
            else
            {
                start_gpu[out_start+cur_start+j]=start_gpu[out_start+prev_start+j];
                x_gpu[out_start+cur_start+j]=x_gpu[out_start+prev_start+j];
                y_gpu[out_start+cur_start+j]=y_gpu[out_start+prev_start+j]+1;
            }
        }

        if(tid==0)
        {

            if(cost_gpu[out_start+cur_start]<small)
            {
                scale_factor=(i-qline+2-start_gpu[out_start+cur_start])*1.0f/qline;
                if(scale_factor>=0.8f&&scale_factor<=1.25f)
                {
                    small=cost_gpu[out_start+cur_start];
                    //start_pos=cur_start_gpu[out_start];
                    //end_pos=i-qline+1;
                }
            }

            /*
            if(bid==TEST_SEQ)
            {
                dist_gpu[i-qline]=cost_gpu[out_start+cur_start];
            }
            */

            inner_start++;
            pprev_start=(inner_start%3)*qline;
            prev_start=((inner_start+1)%3)*qline;
            cur_start=((inner_start+2)%3)*qline;

            end--;
        }

        __syncthreads();
    }

    if(tid==0)
    {
        small_gpu[bid]=small;
    }
}
